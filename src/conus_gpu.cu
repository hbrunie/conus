#include "hip/hip_runtime.h"
#include "conus_gpu.h"

#include <iostream>
#include <Random123/threefry.h>
#include <Random123/ReinterpretCtr.hpp>

#include "example_seeds.h"
#include "util_cuda.h"


using namespace r123;
using namespace std;

void deleteRandomsGPU(double * arr)
{
    CHECKCALL(hipFree(arr));
}

// TODO: will need to figure out what to do with the generate1 VF
// TODO: this will only work for buf_ptr < 4
__device__ double ConusUniformGPU::get()
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    double elt = buf_d[tid+4*buf_ptr[tid]];
    buf_ptr[tid] ++;
    return elt;
}

__device__ void ConusUniformGPU::fill_buf_d()
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    buf_ptr[tid] = 0;
    // uniform_ct_gpu<double>(ulseed, buf_d);
    // typedef Threefry4x64 G;
    union {
        G::ctr_type c;
        long4 i;
    }u;
    int n_cycle = (int)(_N/_nthreads+1) ;
    for (int i_cycle=0; i_cycle<n_cycle; ++i_cycle){
        int idx = 4*tid + i_cycle*_nthreads;
        // Don't advance the RNG if not going to use result
        if (idx + 3 < 4*_N) {
            G rng;
            G::key_type k = {{tid, ulseed}};
            G::ctr_type c = {{}};
            // Grab previous chunck's output state
            if (i_cycle > 0 ) c = buf_state[tid - (i_cycle-1)*_nthreads];

            c.incr();
            u.c = rng(c, k);

            buf_d[idx]   = ((double)((uint64_t)u.i.x))/((double)ULONG_MAX);
            buf_d[idx+1] = ((double)((uint64_t)u.i.y))/((double)ULONG_MAX);
            buf_d[idx+2] = ((double)((uint64_t)u.i.z))/((double)ULONG_MAX);
            buf_d[idx+3] = ((double)((uint64_t)u.i.w))/((double)ULONG_MAX);

            buf_state[tid+i_cycle*_nthreads] = c;
        }
    }
}

__host__ void ConusUniformGPU::initialize()
{
    // TODO: this shouldn't go into the constructor, but we should add
    // a call-guard to prevent repreated calls
    size_t rn_size     = 4*_N * sizeof(double);
    size_t state_size  = _N * sizeof(G::ctr_type);
    size_t ptr_size    = _N * sizeof(int);

    CHECKCALL(hipMalloc(& buf_d, rn_size));
    CHECKCALL(hipMalloc(& buf_state, state_size ));
    CHECKCALL(hipMalloc(& buf_ptr, ptr_size));
    buf_h = (double * ) malloc(4*_N*sizeof(double));
};

// TODO: build destructor to safely free arrays

__host__ void ConusUniformGPU::copyToHost()
{
    // TODO: add checks
    hipMemcpy(buf_h, buf_d, 4*_N*sizeof(double),
               hipMemcpyDeviceToHost);
  _p = -1;
}

// __host__ double ConusUniformGPU::generate1() {
__host__ double ConusUniformGPU::operator()()
{
    _p++;
    if (_p < 4*_N) return buf_h[_p];
    // TODO: figure out what should happen here?
    return -1.;
}


// Entry point
__global__ void generateOnDevice_kernel(ConusUniformGPU * ud_device)
{
    ud_device->fill_buf_d();
}

void generateOnDevice(ConusUniformGPU * ud_host, ConusUniformGPU * ud_device)
{
    unsigned threads_per_block = THREADS_PER_BLOCK;
    // assert(ud.N()%THREADS_PER_BLOCK == 0);
    unsigned blocks_per_grid   = ud_host->N() / threads_per_block;
    generateOnDevice_kernel<<<blocks_per_grid, threads_per_block>>>(ud_device);
}

ConusUniformGPU * sendToDevice(ConusUniformGPU * ud_host)
{
    ConusUniformGPU * ud_device;
    // TODO: add checks
    hipMalloc(& ud_device, sizeof(ConusUniformGPU));
    hipMemcpy(ud_device, ud_host, sizeof(ConusUniformGPU),
               hipMemcpyHostToDevice);
    return ud_device;
}
